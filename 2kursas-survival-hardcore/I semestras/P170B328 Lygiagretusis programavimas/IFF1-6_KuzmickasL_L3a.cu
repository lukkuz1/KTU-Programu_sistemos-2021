#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <nlohmann/json.hpp>
#include <iomanip> 
#include <locale>
#include <codecvt>

using json = nlohmann::json;

#define THREAD_COUNT 64
#define BLOCK_COUNT 2

struct Student {
    std::string name;
    int year;
    float grade;
};

bool compareStudentsByGrade(const Student& a, const Student& b) {
    return a.grade > b.grade;
}

void writeOutputToFile(const std::string& filename, const Student* h_students, const bool* h_filter, int dataSize) {
    std::ofstream outputFile(filename);
    if (outputFile.is_open()) {
        outputFile << "Filtruoti duomenys" << std::endl;
        outputFile << "\n";
        std::vector<Student> sortedStudents;
        for (int i = 0; i < dataSize; ++i) {
            if (h_filter[i]) {
                sortedStudents.push_back(h_students[i]);
            }
        }
        std::sort(sortedStudents.begin(), sortedStudents.end(), compareStudentsByGrade);
        for (const auto& student : sortedStudents) {
            if (student.grade > 75 && student.grade < 80) {
                outputFile << student.name << " C" << std::endl;
            }
            else if (student.grade >= 80 && student.grade < 90) {
                outputFile << student.name << " B" << std::endl;
            }
            else if (student.grade >= 90) {
                outputFile << student.name << " A" << std::endl;
            }
        }
        outputFile.close();
    }
    else {
        std::cerr << "Error opening output file: " << filename << std::endl;
    }
}


void parseJson(const std::string& filename, Student* students, int* dataSize) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    json jsonData;
    file >> jsonData;
    if (!jsonData.is_object() || !jsonData.contains("students") || !jsonData["students"].is_array()) {
        std::cerr << "Error: 'students' array not found in JSON file" << std::endl;
        exit(EXIT_FAILURE);
    }
    auto studentsArray = jsonData["students"];
    *dataSize = std::min(static_cast<int>(studentsArray.size()), *dataSize);
    for (int i = 0; i < *dataSize; ++i) {
        auto studentJson = studentsArray[i];
        if (!studentJson.contains("name") || !studentJson["name"].is_string() ||
            !studentJson.contains("year") || !studentJson["year"].is_number() ||
            !studentJson.contains("grade") || !studentJson["grade"].is_number()) {
            std::cerr << "Error: Invalid data format in JSON file" << std::endl;
            exit(EXIT_FAILURE);
        }
        students[i].name = studentJson["name"].get<std::string>();
        students[i].year = studentJson["year"].get<int>();
        students[i].grade = studentJson["grade"].get<float>();
    }
}


__global__ void filterStudents(const Student* inputStudents, bool* outputFilter, int dataSize, float threshold) {

    int elementsPerBlock = dataSize / BLOCK_COUNT - (dataSize / BLOCK_COUNT % THREAD_COUNT);
    int start = elementsPerBlock * blockIdx.x;
    int end = start + elementsPerBlock;
    if (blockIdx.x == BLOCK_COUNT - 1) {
        end = dataSize;
    }
    for (int i = start + threadIdx.x; i < end; i += THREAD_COUNT) {
        if (i % THREAD_COUNT == threadIdx.x) {
            outputFilter[i] = inputStudents[i].grade >= threshold;
        }
    }
}

int main() {
    const std::string filename = "L3.json";
    const int maxStudents = 250;
    int dataSize = maxStudents;
    const int blockSize = 32;
    const int gridSize = (dataSize + blockSize - 1) / blockSize;
    Student h_students[maxStudents];
    parseJson(filename, h_students, &dataSize);
    std::cout << "Skaitomas json failas: " << filename << std::endl;
    Student* d_students;
    bool* d_filter;
    hipMalloc((void**)&d_students, dataSize * sizeof(Student));
    hipMalloc((void**)&d_filter, dataSize * sizeof(bool));
    hipMemcpy(d_students, h_students, dataSize * sizeof(Student), hipMemcpyHostToDevice);
    // filtravimo sąlyga
    float threshold = 70.0;
    filterStudents << <gridSize, blockSize >> > (d_students, d_filter, dataSize, threshold);
    hipDeviceSynchronize();
    bool h_filter[maxStudents];
    hipMemcpy(h_filter, d_filter, dataSize * sizeof(bool), hipMemcpyDeviceToHost);
    writeOutputToFile("islaike_studentai.txt", h_students, h_filter, dataSize);
    std::cout << "Duomenys sėkmingai sufiltruoti" << std::endl;
    hipFree(d_students);
    hipFree(d_filter);
    std::cout << "Atlaisvinama atmintis " << std::endl;
    return 0;
}
